#include "hip/hip_runtime.h"
#include "CudaIncludesAndMacros.cuh"
#include "PlotPointsBackwardsPasses.cuh"
#include "cuda_debug_utils.cuh"

template <typename camera_type_partial>
__global__ void backwards_points(const camera_type_partial camera, int ndim, float* point_grad, const float* point_data, int num_points, const float* plot, const float* plot_grad, const float* plot_weights) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < num_points) {
        int ids = idx * (3 + ndim);
        //NOTE: d is calculated both here AND in the forward pass and could theoretically be saved.
        ScreenCoordsWithDepth d = camera.mapToScreenCoords(make_float4(point_data[ids + 0], -point_data[ids + 1], point_data[ids + 2], 1));
        if (d.valid) {
            int pixel = d.coords.x + d.coords.y * camera.w;
            float depth = d.depth;
            float surface_depth = plot[pixel * (ndim + 1) + ndim];
            //I should probably move the depth test to a shared header, as well as a function for weight (which is currently always 1)
            if (surface_depth * (1 + 0.001) < depth || plot_weights[pixel]<=0)return;
            float weight_fraction = 1 / plot_weights[pixel];
            const float* grad_start = &plot_grad[pixel * (ndim + 1)];
            float* point_grad_start = &(point_grad[ids + 3]);
            for (int i = 0; i < ndim; ++i) {
                //Do not ask me wy this has to be a minus. I do not know.
                //point_grad_start[i] -= grad_start[i] * weight_fraction;
                point_grad_start[i] += grad_start[i] * weight_fraction;
            }
            //todo: point position refinement, somehow.
        }
    }
}

template <typename camera_type_partial>
__global__ void backwards_environment_v2(const camera_type_partial camera, int ndim, float* environment_grad, int environment_resolution, const float* plot_weights, const float* plot_grad) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < camera.h && idy < camera.w) {
        int ids = (idy + idx * camera.w);
        int ids_m = (idy + idx * camera.w) * (ndim + 1);
        if (plot_weights[ids] <= 0) {//weight = 0 would mean no points landed there
            float3 direction = camera.direction_for_pixel(make_int2(idy, idx));
            unsigned int adress = (ndim+1)*pixel_from_cubemap_coords(environment_resolution, cubemap_coords(environment_resolution, direction));
            //Maybe I should use supersampling? There seems to be somme weird static that rendering at lower resolutions seems to cause.
            for (int i = 0; i < ndim + 1; ++i) {
                atomicAdd(&environment_grad[adress + i], plot_grad[ids_m + i]);
            }
            //atomicAdd(&environment_grad[adress + ndim], plot_grad[ids + ndim]);
            //NOTE: this represents depth, and was separated for no good reason.
        }
    }
}


template <typename camera_type_partial>
hipError_t backwards_environment_for_camera_v2(const camera_type_partial& camera, int ndim, 
    void* environment_grad, int environment_resolution,
    const void* plot_weights, const void* plot_grad) {
    hipError_t cudaStatus;
    backwards_environment_v2 BEST_2D_KERNEL(camera.h, camera.w) (camera, ndim, (float*)environment_grad, environment_resolution, (float*)plot_weights, (float*)plot_grad);
    AFTER_FUNCTION_CALL_CHECK();
Error:
    return cudaStatus;
}

template <typename camera_type_partial>
hipError_t backwards_points_for_camera_v2(const camera_type_partial& camera, int ndim,
    const void* point_grad, const void* point_data, int num_points,
    const void* plot, const void* plot_grad, const void* plot_weights) {
    hipError_t cudaStatus;
    backwards_points BEST_LINEAR_KERNEL(num_points) (camera, ndim, (float*)point_grad, (const float*)point_data, num_points, (const float*)plot, (const float*)plot_grad, (const float*)plot_weights);
    AFTER_FUNCTION_CALL_CHECK();
Error:
    return cudaStatus;
}



template <typename camera_type_partial>
hipError_t backwards_for_camera_v2(const camera_type_partial& camera, int ndim,
    const void* point_data, void* point_grad, int num_points,
    const void* environment, void* environment_grad, int environment_resolution,
    const void* plot, const void* plot_weights, const void* plot_grad){
    hipError_t cudaStatus;
    cudaStatus = backwards_environment_for_camera_v2(camera, ndim, environment_grad, environment_resolution, plot_weights, plot_grad);
    STATUS_CHECK();
    cudaStatus = backwards_points_for_camera_v2(camera, ndim, point_grad, point_data, num_points, plot, plot_grad, plot_weights);
    STATUS_CHECK();
    ENSURE_SYNC();
Error:
    return cudaStatus;
}


hipError_t PlotPointsBackwardsPass_v2(const std::shared_ptr<CameraDataItf> camera, int ndim,
    const void* point_data, void* point_grad, int num_points,
    const void* environment, void* environment_grad, int environment_resolution,
    const void* plot, const void* plot_weights, const void* plot_grad)
{
    //todo? delete environment from signature and calls since it is not used?
    hipError_t cudaStatus;
    int h = camera->get_height();
    int w = camera->get_width();
    switch (camera->type())
    {
    case INTERACTIVE:
        //this branch shouldn't even be used, as in theory it would mean an interactive camera is being used for training data.
        throw "Can't use interactive camera for backwards pass.";
        /*cudaStatus = backwards_for_camera_v2(((InteractiveCameraData&)(*camera)).prepareForGPU(w, h), ndim,
            point_data, point_grad, num_points,
            environment, environment_grad, environment_resolution, plot, plot_weights, plot_grad);
        STATUS_CHECK();*/
        break;
    case PINHOLE_PROJECTION:
        cudaStatus = backwards_for_camera_v2(((PinholeCameraData    &)(*camera)).prepareForGPU(w, h), ndim,
            point_data, point_grad, num_points,
            environment, environment_grad, environment_resolution, plot, plot_weights, plot_grad);
        STATUS_CHECK();
        break;
    default:
        std::cerr << "INVALID CAMERA FOUND AT " << __LINE__ << " of " << __FILE__ << '\n';
        break;
    }
Error:
    //TODO: implement
    return cudaStatus;
}
