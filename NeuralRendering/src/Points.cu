#include "hip/hip_runtime.h"
#include "Points.hpp"
#include <fstream>
#include <vector>
#include "CudaIncludesAndMacros.cuh"
#include "stream_binary_utils.hpp"
#include "CudaMath.cuh"
#if __CUDA_ARCH__ < 600
static_assert(false, "only 6.0 or greater currently supported.")
#endif
hipError_t GPUPoints::writeToFile(const char* path, int vers)
{
	hipError_t cudaStatus = hipSuccess;
	switch (vers) {
	case 0x600:case 0x610: {
		//Positions file
		{
			size_t num_bytes = sizeof(float)* 3 * num_entries;
			std::unique_ptr<float> fvalues = std::unique_ptr<float>((float*)malloc(num_bytes));
			if (fvalues == NULL) return hipError_t::hipErrorOutOfMemory;
			cudaStatus = hipMemcpy(fvalues.get(), position_memory_start, num_bytes, hipMemcpyKind::hipMemcpyDeviceToHost);
			STATUS_CHECK();
			std::ofstream f{ path,std::ios::binary };
			f.write((char*)fvalues.get(), num_bytes);
		}
		//Colors file
		{
			size_t num_bytes = sizeof(byte) * 3 * num_entries;
			std::unique_ptr<byte> cvalues = std::unique_ptr<byte>((byte*)malloc(num_bytes));
			if (cvalues == NULL) return hipError_t::hipErrorOutOfMemory;
			cudaStatus = hipMemcpy(cvalues.get(), color_memory_start, num_bytes, hipMemcpyKind::hipMemcpyDeviceToHost);
			STATUS_CHECK();
			std::ofstream f{ std::string(path) + ".colors",std::ios::binary };
			f.write((char*)cvalues.get(), num_bytes);
		}
		//environment file
		if (environment != nullptr) {
			int size = environment->resolution * environment->resolution * (environment->data_is_bytes ? sizeof(char) : sizeof(float)) * environment->dimensions * 6;
			auto to_output = std::unique_ptr<char>((char*)malloc(size));
			hipMemcpy(to_output.get(), environment->data, size, hipMemcpyDeviceToHost);
			std::ofstream f{ std::string(path) + ".environment",std::ios::binary };
			int tmp = environment->resolution;
			f.write((char*)&tmp, sizeof(tmp));
			tmp = (environment->data_is_bytes ? 0 : 1);
			f.write((char*)&tmp, sizeof(tmp));
			f.write((char*)to_output.get(), size);
		}
	}break;
	default: {
		std::cerr << "Point cloud not saved! Version " << std::hex << vers << std::dec << " not supported!\n";
	}break;
	}
Error:
	return cudaStatus;
}

hipError_t GPUPoints::read(const char* path, GPUPoints& dest, int vers)
{
	std::shared_ptr<Environment> environment = nullptr;
	hipError_t cudaStatus = hipSuccess;
	std::vector<float> positions;
	std::vector<unsigned char> colors;
	switch (vers)
	{
	case 0x500: {
		std::ifstream f{ path };
		if (f.fail())return hipError_t::hipErrorFileNotFound;
		int size = -1, c = 0;
		if (!(f >> size >> c))return hipError_t::hipErrorUnknown;
		if (size != -1) { positions.reserve(size); colors.reserve(size); }
		if (c == 0) { c = 1; }
		float x, y, z; int r, g, b;

		while (f >> x >> y >> z >> r >> g >> b)
		{
			positions.push_back(x);
			positions.push_back(y);
			positions.push_back(z);
			colors.push_back(clamp(r / c, 0, 255));
			colors.push_back(clamp(g / c, 0, 255));
			colors.push_back(clamp(b / c, 0, 255));
		}
		if (f.fail() && !f.eof())return hipError_t::hipErrorUnknown;
	}break;
	case 0x600:case 0x610: {
		std::ifstream f1{ path, std::ios::binary };
		if (f1.fail())return hipError_t::hipErrorFileNotFound;
		readBinaryIntoArray(positions, f1, -1);
		f1.close();
		std::ifstream f2{ std::string(path) + ".colors", std::ios::binary };
		if (f2.fail())return hipError_t::hipErrorFileNotFound;
		readBinaryIntoArray(colors, f2, -1);
		//There may also be a .environment file
		std::ifstream f3{ std::string(path) + ".environment", std::ios::binary };
		if (!f3.fail()) {
			int resolution = readOneBinary<int>(f3);
			bool as_bytes = (0==readOneBinary<int>(f3));
			environment = std::make_shared<Environment>(resolution,as_bytes);
			if (as_bytes) {
				std::vector<unsigned int> bytes;
				readBinaryIntoArray(bytes, f3, -1);
				const int expected = sizeof(uchar4) * environment->resolution * environment->resolution * 6;
				if (bytes.size() * sizeof(unsigned int) != expected) {
					std::cerr << "Byte environment invalid\n";
					return hipError_t::hipErrorUnknown;
				}
				hipMemcpy(environment->data, &bytes[0], expected, hipMemcpyHostToDevice);
			} else {
				std::vector<float> floats;
				readBinaryIntoArray(floats, f3, -1);
				const int expected = sizeof(float4) * environment->resolution * environment->resolution * 6;
				if (floats.size() * sizeof(float) != expected) {
					std::cerr << "Float environment invalid\n";
					return hipError_t::hipErrorUnknown;
				}
				hipMemcpy(environment->data, &floats[0], expected, hipMemcpyHostToDevice);
			}
		}
	}break;
	default:
		break;
	}
#ifdef _DEBUG
	if (positions.size() != colors.size()) {
		std::cerr << "Not the same number of points as colors\n";
		return hipError_t::hipErrorUnknown;
	}
#endif
	positions.shrink_to_fit();
	colors.shrink_to_fit();
	int sz = (int) positions.size();
	void* gpu_pos;
	void* gpu_col;
	cudaStatus = hipMalloc(&gpu_pos, sz * sizeof(float));
	if (cudaStatus != hipSuccess) { hipFree(gpu_pos); }
	STATUS_CHECK();
	cudaStatus = hipMemcpy(gpu_pos, positions.data(), sz * sizeof(float),hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { hipFree(gpu_pos); }
	STATUS_CHECK();
	cudaStatus = hipMalloc(&gpu_col, sz * sizeof(char));
	if (cudaStatus != hipSuccess) { hipFree(gpu_pos); }
	STATUS_CHECK();
	cudaStatus = hipMemcpy(gpu_col, colors.data(), sz * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { hipFree(gpu_pos); hipFree(gpu_col); }
	STATUS_CHECK();
	dest = GPUPoints(gpu_pos, gpu_col, sz / 3, true, environment);
Error:
	return cudaStatus;
}
GPUPoints::~GPUPoints()
{
	if (should_free) {
		hipFree(color_memory_start);
		hipFree(position_memory_start);
	}
}

Environment::Environment(int resolution, bool as_bytes, int dimensions) :resolution{ resolution }, data_is_bytes{ as_bytes }, dimensions{ dimensions }
{
	if (data_is_bytes) {
		grad = NULL;
		hipError_t error = hipMalloc(&data, sizeof(char) * dimensions * resolution * resolution * 6);//6 faces on a cubemap.
		if (error != hipSuccess) { this->resolution = 0; data = 0; }
		error = hipMemset(data, 0, sizeof(char) * dimensions * resolution * resolution * 6);//6 faces on a cubemap.
		if (error != hipSuccess) { this->resolution = 0; data = 0; }
	} else {
		grad = NULL;
		hipError_t error = hipMalloc(&data, sizeof(float) * dimensions * resolution * resolution * 6);//6 faces on a cubemap.
		if (error != hipSuccess) { this->resolution = 0; data = 0; }
		error = hipMemset(data, 0, sizeof(float) * dimensions * resolution * resolution * 6);//6 faces on a cubemap.
		if (error != hipSuccess) { this->resolution = 0; data = 0; }
	}
}
hipError_t Environment::ensure_grad() {
	if (grad == NULL) {
		hipError_t error = hipMalloc(&grad, sizeof(float) * dimensions * resolution * resolution * 6);//6 faces on a cubemap.
		zero_grad();
		return error;
	}
	return hipSuccess;
}

hipError_t Environment::zero_grad()
{
	if (grad != NULL)
		return set_all_to_zeros(grad, sizeof(float) * dimensions * resolution * resolution * 6);
	return hipError_t::hipErrorIllegalState;
}

#include "cuda_debug_utils.cuh"

hipError_t Environment::merge_grad()
{
	if (grad != NULL) {
		//printGPUMemoryCoord<float>(grad, 4 * resolution * resolution * 6, [](float x) {return x!=0.0f; }, 4, 4 * resolution, 4 * resolution * resolution);
		return apply_gradient(data, grad, (size_t) dimensions * resolution * resolution * 6, this->data_is_bytes);
	}
	return hipError_t::hipErrorIllegalState;
}


Environment::~Environment()
{
	if (grad != NULL) {
		hipFree(grad);
	}
	if (data != NULL) {
		hipFree(data);
	}
}
