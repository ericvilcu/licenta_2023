#include "hip/hip_runtime.h"
#ifndef STRUCTURAL_REFINEMENT
#define STRUCTURAL_REFINEMENT 0
#endif
#ifndef CAM_TYPE
#define CAM_TYPE 0
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef CAMERA_GRAD
#define CAMERA_GRAD 0
#endif
#include <hip/hip_runtime.h>
//TODO: better split functions into their own subkernels

float __hdfi__ test_depth(float my_depth, float min_depth){
    if(min_depth * (1 + 0.001) < my_depth) return -1;
    return 1;
}
//Step 1 of plotting:
void __global__ translateKernelCharToSurface(hipSurfaceObject_t output, const uchar4* colors, const int h, const int w, const int hd = 0, const int wd = 0) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < h && idy < w) {
        surf2Dwrite(colors[idy+wd + (hd+idx) * w], output, idy * sizeof(uchar4), idx, hipBoundaryModeClamp);
    }
}
//__global__ clear(float*color,float*weight,int mx,int my){
//    int idx = threadIdx.x + blockDim.x * blockIdx.x;
//    int idy = threadIdx.y + blockDim.y * blockIdx.y;
//    if (idx < h && idy < w) {
//    }
//}

void __global__ determine_depth(float* output, const float* points, int num_points, const float* camera_raw_data){
    Camera camera{camera_raw_data};
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < num_points) {
        //float k=1.0;
        //float x=k/0.;
        int w=(int)camera.w;
        //int h=(int)camera.h;
        // X,Y,Z, (NDIM 'colors')
        int pos = (3 + NDIM) * idx;
        float3 position = make_float3(points[pos], -points[pos + 1], points[pos + 2]);
        auto rez = camera.mapToScreenCoords(position);
        int2 screen_coords = rez.coords;
        float depth = rez.depth;
        if (!rez.valid)return;
        float* pixel_depth = &output[(NDIM + 1) * (screen_coords.x + screen_coords.y * w) + NDIM];
        atomicMin((int*)pixel_depth, *(int*)(float*)&depth);
        static_assert(sizeof(int) == sizeof(float), "Here so the atomicMin hack above works properly. If this does not work, some other suitable replacement for it should be found, potentially using atomicCAS");
    }
}

void __global__ plot(float* output, float* weights, const float* points, int num_points, const float* camera_raw_data/*, int32_t* dbg0 = NULL*/){
    Camera camera{camera_raw_data};
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < num_points) {
        int w=(int)camera.w;
        //int h=(int)camera.h;
        // X,Y,Z, (NDIM 'colors')
        int pos = (3 + NDIM) * idx;
        float4 position = make_float4(points[pos], -points[pos + 1], points[pos + 2], 1);
        //const float* this_point_data = &point_data[ids];
        //ScreenCoordsWithDepth d = camera.mapToScreenCoords(make_float4(this_point_data[0], -this_point_data[1], this_point_data[2], 1));
        auto rez = camera.mapToScreenCoords(position);
        int2 screen_coords = rez.coords;
        //if(dbg0!=NULL){dbg0[2*idx]=screen_coords.x;dbg0[2*idx+1]=screen_coords.y;}
        float depth = rez.depth;
        if (!rez.valid)return;
        const float pixel_depth = output[(NDIM + 1) * (screen_coords.x + screen_coords.y * w) + NDIM];
        //todo: move depth test to a header or other special function
        float weight = test_depth(depth,pixel_depth);
        if (weight<=0)
            return;
        #pragma unroll
        for (int i = 0; i < NDIM; ++i) {
            float* p = &output[(NDIM + 1) * (screen_coords.x + screen_coords.y * w) + i];
            atomicAdd(p, points[pos + 3 + i]);
        }
        float* pixel_color_weight_location = &weights[screen_coords.x + screen_coords.y * w];
        atomicAdd(pixel_color_weight_location, weight);
    }
}

//todo: plot environment simultaniously
void __global__ bundle(float* plot, float* weights, float* environment_data, const int h, const int w,const float* camera_raw_data){
    Camera camera{camera_raw_data};
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < h && idy < w) {
        int ids = idy + idx * w;
        if (weights[ids] > 0) {
            int ids_m = ids * (NDIM + 1);
            float local_weight = weights[ids];
            for (int i = 0; i < NDIM; ++i)
                plot[ids_m + i] = plot[ids_m + i] / local_weight;
        } else {
            int ids_m = ids * (NDIM + 1);
            float3 direction = camera.direction_for_pixel(make_float2(idx+0.5f,idy+0.5f));
            sample_environment(&plot[ids_m],environment_data,direction);
        }
    }
}
//todo? switch to normal arrays for camera? as in, a[20], so it passes all elements but not by reference.
void __global__ backward(float* camera_gradient, const float* camera_data, float* point_grad, const float* point_data, int num_points, const float* plot, const float* plot_grad, const float* plot_weights/*,int32_t* dbg0 = NULL*/){
    Camera camera{camera_data};
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < num_points) {
        int ids = idx * (3 + NDIM);
        int w=(int)camera.w;
        int h=(int)camera.h;
        //NOTE: d is calculated both here AND in the forward pass and could theoretically be saved.
        const float* this_point_data = &point_data[ids];
        ScreenCoordsWithDepth d = camera.mapToScreenCoords(make_float4(this_point_data[0], -this_point_data[1], this_point_data[2], 1));
        if (d.valid) {
            int pixel = d.coords.x + d.coords.y * w;
            //if(dbg0!=NULL){dbg0[2*idx]=d.coords.x;dbg0[2*idx+1]=d.coords.y;}
            float depth = d.depth;
            float pixel_depth = plot[pixel * (NDIM + 1) + NDIM];
            //I should probably move the depth test to a shared header, as well as a function for weight (which is currently always 1)
            float weight = test_depth(depth,pixel_depth);
            if (weight<=0)
                return;
            float weight_fraction = 1 / weight;
            const float* grad_start = &plot_grad[pixel * (NDIM + 1)];
            float* point_grad_start = &(point_grad[ids]);
            float* point_color_grad_start = &(point_grad[ids + 3]);
            for (int i = 0; i < NDIM; ++i) {
                point_color_grad_start[i] += grad_start[i] * weight_fraction;//+=0
            }
            //estimate for point position refinement, based on: https://arxiv.org/pdf/2110.06635.pdf
            //Note: the edges of the image are iffy, so I exclude them.
            if(STRUCTURAL_REFINEMENT!=0 && d.coords.x>0 && d.coords.y>0 && d.coords.x<w-1 && d.coords.y<h-1)//todo: verify the implementation at the camera level.
            {
                auto compute_position_grad = [&](int pixel) -> float
                {
                    const float* pixel_data = &plot[pixel * (NDIM + 1)];
                    float pixel_depth = pixel_data[NDIM];
                    const float* pixel_grad = &plot_grad[pixel * (NDIM + 1)];
                    float pixel_weight = plot_weights[pixel];
                    float d=0.0;
                    if (pixel_weight <= 0) {
                        //overwrites background
                        for (int i = 0; i < NDIM; ++i) {
                            d += (this_point_data[3 + i] - pixel_data[i]) * pixel_grad[i];
                        }
                    } else{
                        float weight = test_depth(depth,pixel_depth);
                        if (weight<=0) {
                            //does nothing, as it would be far behind other points, not contribuiting.
                        } else {
                            //blended with other points
                            float new_inv_weight = 1 / (pixel_weight+1);
                            for (int i = 0; i < NDIM; ++i) {
                                d += new_inv_weight * (this_point_data[3 + i] - pixel_weight * pixel_data[i]) * pixel_grad[i];
                            }
                        }
                    }
                    return d;
                };
                int pixelX0 = (d.coords.x-1) + d.coords.y * w;
                int pixelX1 = (d.coords.x+1) + d.coords.y * w;
                int pixelY0 = d.coords.x + (d.coords.y-1) * w;
                int pixelY1 = d.coords.x + (d.coords.y+1) * w;
                float grad_X = 0.5f * (-compute_position_grad(pixelX0) + compute_position_grad(pixelX1));
                float grad_Y = 0.5f * (-compute_position_grad(pixelY0) + compute_position_grad(pixelY1));
                float3 pixel_direction = camera.direction_for_pixel(make_float2(d.coords.x+0.5f, d.coords.y+0.5f));
                float3 wanted_direction = camera.direction_for_pixel(make_float2(d.coords.x + grad_X+0.5f, d.coords.y + grad_Y+0.5f));
                float3 gradient_screen_space = make_float3((pixel_direction.x - wanted_direction.x) * -depth, (pixel_direction.y - wanted_direction.y) * -depth, (pixel_direction.z - wanted_direction.z) * -depth);
                point_grad_start[0] += gradient_screen_space.x;
                point_grad_start[1] += gradient_screen_space.y;
                point_grad_start[2] += gradient_screen_space.z;
/*#ifdef CAMERA_GRAD
#pragma message("Warn: Only camera position gradient is implemented, rotation and intrinsics are not.")
                atomicAdd(&camera_gradient[4+9+0],-gradient_screen_space.x);
                atomicAdd(&camera_gradient[4+9+1],-gradient_screen_space.y);
                atomicAdd(&camera_gradient[4+9+2],-gradient_screen_space.z);
#endif*/
            }
        }
    }
}

//Note: plot currently unused
void __global__ backward_pixel(float*cam_data_grad,
    const float*camera_data,
    const float*plot, const float*plot_grad, const float*weights,
    int h,int w,
    float* environment_grad,const float*environment){
    Camera camera{camera_data};
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < h && idy < w) {
        int ids = idy + idx * w;
        if (weights[ids] > 0) {
            int ids_m = ids * (NDIM + 1);
            float3 direction = camera.direction_for_pixel(make_float2(idx+0.5f, idy+0.5f));
            backward_environment(plot_grad+ids_m, environment, environment_grad, direction);
        }
    }
}